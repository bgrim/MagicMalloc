#include<stdio.h>
#include<hip/hip_runtime.h>

struct memoryPointer{
  memoryPointer *ptr;
  unsigned size;
  unsigned *data;
};

typedef struct memoryPointer MemoryPointer;

static MemoryPointer base;
static MemoryPointer *freep = NULL;

void fastAddList(MemoryPointer *bp){
  MemoryPointer *p;

  for(p = freep; !(bp->data > p->data && bp->data < (p->ptr)->data); p = p->ptr)
    if(p->data >= (p->ptr)->data && (bp->data > p->data || bp->data < (p->ptr)->data)) 
      break;

  if( ((MemoryPointer *) (((char *)bp->data) + bp->size)) == p->ptr){
    bp->size += (p->ptr)->size;
    bp->ptr = (p->ptr)->ptr;
    hipMemcpy(bp->data, &bp->size, sizeof(unsigned), hipMemcpyHostToDevice);
    free(p->ptr);
  }else
    bp->ptr = p->ptr;

  if(  ((MemoryPointer *) (((char *)p->data) + p->size)) == bp){
    p->size += bp->size;
    p->ptr = bp->ptr;
    hipMemcpy(p->data, &p->size, sizeof(unsigned), hipMemcpyHostToDevice);
    free(bp);
  }else
    p->ptr = bp;
  
  freep = p;
}

void fastFree(void *loc){
  loc = ((void *)(((char *)loc)-sizeof(unsigned)));
  MemoryPointer *v = (MemoryPointer *) malloc(sizeof(MemoryPointer));
  hipMemcpy(&v->size, loc, sizeof(unsigned), hipMemcpyDeviceToHost);
  v->data = (unsigned *) loc;
  fastAddList(v);
}


static MemoryPointer *morecore(unsigned nu){
  void *cp;
  MemoryPointer *up = (MemoryPointer *)malloc(sizeof(MemoryPointer));
  if (nu < 1048576) nu = 1048576;
  hipMalloc(&cp, nu);

  up->data = (unsigned *)cp;

  up->size = nu;
  hipMemcpy(cp,&(up->size),sizeof(unsigned),hipMemcpyHostToDevice);

  fastAddList(up);
  return freep;
}



void *fastMalloc(unsigned nbytes){
  MemoryPointer *p, *prevp;
  if ((prevp = freep)==NULL){
    base.ptr = freep = prevp = &base;
    base.size = 0;
  }
  nbytes+=sizeof(unsigned);
  char *loc;
  for(p = prevp->ptr; ;prevp = p, p = p->ptr){
    if(p->size >= nbytes){
      if(p->size == nbytes){
        prevp->ptr = p->ptr;
        loc = (char *) p->data;
        free(p);
      }else{
        p->size -= nbytes;
        loc =((char *) p->data)+p->size;
      }
      freep = prevp;
      hipMemcpy(loc,&nbytes,sizeof(unsigned),hipMemcpyHostToDevice);
      return (void *)(loc+sizeof(unsigned));
    }
    if (p == freep)
      if((p = morecore(nbytes))==NULL)
        return NULL;
  }
}


int main(int argc, char **argv){
  void *v;
  int i, cap=0;
  if(argc>1)cap=atoi(argv[1]);
  for(i=0; i<cap; i++){
    v = fastMalloc(1);
  }
}
